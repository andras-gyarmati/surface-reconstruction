#include "hip/hip_runtime.h"

#include "hipsolver.h"
#include "hiprand/hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include <exception>
#include <random>

//__device__ void printMat(float** mat, int n, int m)
//{
//	for (int i = 0; i < n; i++)
//	{
//		for (int j = 0; j < m; j++)
//		{
//			printf("%f ", mat[i][j]);
//		}
//		printf("\n");
//	}
//}

class CudaException : public std::exception
{
	virtual const char* what() const throw()
	{
		return "GPUassert fail";
	}
};

#define checkCudaCall(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		throw CudaException();
	}
}

__device__ float distance(float* model, float* p)
{
	return fabsf(model[0] * p[0] + model[1] * p[1] + model[2] * p[2] + model[3]);
}

__global__ void countInliers(float* pts, float* eigenvectors, int* count_inliers, float* plane_params, int size, float threshold)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	float A = eigenvectors[id * 16];
	float B = eigenvectors[id * 16 + 1];
	float C = eigenvectors[id * 16 + 2];
	float D = eigenvectors[id * 16 + 3];

	float norm = sqrtf(A * A + B * B + C * C);

	plane_params[id * 4] = A / norm;
	plane_params[id * 4 + 1] = B / norm;
	plane_params[id * 4 + 2] = C / norm;
	plane_params[id * 4 + 3] = D / norm;

	float model[4] = { plane_params[id * 4] , plane_params[id * 4 + 1] , plane_params[id * 4 + 2], plane_params[id * 4 + 3]};
	//float model[3] = { plane_params[id * 3] , plane_params[id * 3 + 1] , plane_params[id * 3 + 2]};

	int count = 0;
	for (int i = 0; i < size; i++)
	{
		float p[3] = { pts[i * 3], pts[i * 3 + 1], pts[i * 3 + 2]};
		if (distance(model, p) < threshold)
			count++;
	}

	count_inliers[id] = count;
}

__global__ void createSystems(float* pts, float* As, float** bs, int seed, hiprandState* rand_state, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	hiprand_init(seed, id, 0, &rand_state[id]);
	int idx1, idx2, idx3;
	idx1 = hiprand_uniform(&(rand_state[id])) * size - 1;	//curand excludes 0.0 and includes 1.0
	idx2 = hiprand_uniform(&(rand_state[id])) * size - 1;
	idx3 = hiprand_uniform(&(rand_state[id])) * size - 1;

	float Aorig[3][4];
	Aorig[0][0] = pts[idx1 * 3];
	Aorig[0][1] = pts[idx1 * 3 + 1];
	Aorig[0][2] = pts[idx1 * 3 + 2];
	Aorig[0][3] = 1;
	Aorig[1][0] = pts[idx2 * 3];
	Aorig[1][1] = pts[idx2 * 3 + 1];
	Aorig[1][2] = pts[idx2 * 3 + 2];
	Aorig[1][3] = 1;
	Aorig[2][0] = pts[idx3 * 3];
	Aorig[2][1] = pts[idx3 * 3 + 1];
	Aorig[2][2] = pts[idx3 * 3 + 2];
	Aorig[2][3] = 1;
	//printf("%d, %d, %d\n", idx1, idx2, idx3);

	//printf("Aorig:\n");
	//for (int i = 0; i < 3; i++)
	//{
	//	for (int j = 0; j < 4; j++)
	//	{
	//		printf("%f ", Aorig[i][j]);
	//	}
	//	printf("\n");
	//}
	
	//printf("At:\n");
	float At[4][3];
	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 3; j++)
		{
			At[i][j] = Aorig[j][i];
		}
	}

	//for (int i = 0; i < 4; i++)
	//{
	//	for (int j = 0; j < 3; j++)
	//	{
	//		printf("%f ", At[i][j]);
	//	}
	//	printf("\n");
	//}

	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			As[id * 16 + j * 4 + i] = 0;

			for (int k = 0; k < 3; k++) {
				//Column major ordering
				As[id * 16 + j * 4 + i] += At[i][k] * Aorig[k][j];
			}
		}
	}

	//printf("A:\n");
	//for (int i = 0; i < 4; i++)
	//{
	//	for (int j = 0; j < 4; j++)
	//	{
	//		printf("%f ", As[id * 16 + i * 4 + j]);
	//	}
	//	printf("\n");
	//}

	//Put them in column major for cublas
	//x, y, z, 1
	//As[id * 16] = pts[idx1 * 3];
	//As[id][1] = pts[idx2 * 3];
	//As[id][2] = pts[idx3 * 3];
	//As[id][3] = pts[idx1 * 3 + 1];
	//As[id][4] = pts[idx2 * 3 + 1];
	//As[id][5] = pts[idx3 * 3 + 1];
	//As[id][6] = pts[idx1 * 3 + 2];
	//As[id][7] = pts[idx2* 3 + 2];
	//As[id][8] = pts[idx3 * 3 + 2];
	////As[id][9] = 1;
	////As[id][10] = 1;
	////As[id][11] = 1;

	////homogeneous system
	//bs[id][0] = -1;
	//bs[id][1] = -1;
	//bs[id][2] = -1;
	//bs[id][3] = 0;
}

hipError_t runRansacGPU(const std::vector<float>& pts, std::vector<float>& plane_params, std::vector<int>& counts, const int& iters, const float& threshold, const int& threadNum)
{
	float* d_pts = 0;
	float* d_plane_params = 0;
	int* d_counts = 0;
	//float** d_As = 0;
	float* d_As = 0;
	float** d_bs = 0;

	int* h_counts = (int*)malloc(sizeof(int) * iters);
	float* h_plane_params = (float*)malloc(sizeof(float) * 4 * iters);
	float** h_As = (float**)malloc(iters * sizeof(float*));
	float** h_bs = (float**)malloc(iters * sizeof(float*));

	hiprandState* d_state;
	hipError_t cudaStatus;

	hipblasHandle_t cublasH = NULL;

	try {
		checkCudaCall(hipSetDevice(0));

		checkCudaCall(hipMalloc((void**)&d_pts, pts.size() * sizeof(float)));
		checkCudaCall(hipMalloc((void**)&d_plane_params, iters * 4 * sizeof(float)));
		checkCudaCall(hipMalloc((void**)&d_counts, iters * sizeof(float)));

		checkCudaCall(hipMalloc(&d_state, iters * sizeof(hiprandState)));
		//checkCudaCall(cudaMalloc((void**)&d_As, iters * sizeof(float*)));
		checkCudaCall(hipMalloc((void**)&d_As, iters * 16 * sizeof(float)));
		checkCudaCall(hipMalloc((void**)&h_As[0], iters * 9 * sizeof(float)));
		checkCudaCall(hipMalloc((void**)&d_bs, iters * sizeof(float*)));
		checkCudaCall(hipMalloc((void**)&h_bs[0], iters * 3 * sizeof(float)));

		for (int i = 1; i < iters; i++)
		{
			h_As[i] = h_As[i - 1] + 9;
			h_bs[i] = h_bs[i - 1] + 3;
		}

		checkCudaCall(hipMemcpy(d_pts, pts.data(), pts.size() * sizeof(float), hipMemcpyHostToDevice));
		//checkCudaCall(cudaMemcpy(d_As, h_As, iters * sizeof(float*), cudaMemcpyHostToDevice));
		checkCudaCall(hipMemcpy(d_bs, h_bs, iters * sizeof(float*), hipMemcpyHostToDevice));

		srand(time(0));
		int seed = rand();

		//createSystems <<< 1, 1 >>> (d_pts, d_As, d_bs, seed, d_state, pts.size() / 3);
		createSystems <<< iters / threadNum, threadNum >>> (d_pts, d_As, d_bs, seed, d_state, pts.size() / 3);

		//////////////////
		hipsolverHandle_t cusolverH = NULL;
		hipStream_t stream = NULL;
		hipsolverSyevjInfo_t syevj_params = NULL;

		const int m = 4;
		const int lda = 4;
		const int batchSize = iters;

		//std::vector<float> A(lda * m * batchSize, 0); /* A = [A0 ; A1] */
		std::vector<float> V(lda * m * iters, 0); /* V = [V0 ; V1] */
		std::vector<float> W(m * iters, 0);       /* W = [W0 ; W1] */
		std::vector<int> info(batchSize, 0);           /* info = [info0 ; info1] */

		float* d_A = nullptr;    /* lda-by-m-by-batchSize */
		float* d_W = nullptr;    /* m-by-batchSize */
		int* d_info = nullptr;    /* batchSize */
		float* d_work = nullptr; /* device workspace for syevjBatched */
		int lwork = 0;            /* size of workspace */

		/* configuration of syevj  */
		const double tol = 1.e-7;
		const int max_sweeps = 15;
		const int sort_eig = 1;                                  /* don't sort eigenvalues */
		const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute eigenvectors */
		const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

		//double* A0 = A.data();
		//double* A1 = A.data() + lda * m;

		hipsolverDnCreate(&cusolverH);
		hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
		hipsolverSetStream(cusolverH, stream);
		hipsolverDnCreateSyevjInfo(&syevj_params);
		hipsolverDnXsyevjSetTolerance(syevj_params, tol);
		hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps);
		hipsolverDnXsyevjSetSortEig(syevj_params, sort_eig);

		//cudaMalloc(reinterpret_cast<void**>(&d_A), sizeof(double) * A.size());
		hipMalloc((void**)&d_W, sizeof(float) * W.size());
		hipMalloc((void**)&d_info, sizeof(int) * info.size());
		//cudaMemcpyAsync(d_A, A.data(), sizeof(double) * A.size(), cudaMemcpyHostToDevice, stream));
		hipsolverDnSsyevjBatched_bufferSize(cusolverH, jobz, uplo, m, d_As, lda, d_W, &lwork, syevj_params, batchSize);
		hipMalloc(reinterpret_cast<void**>(&d_work), sizeof(double) * lwork);
		hipsolverDnSsyevjBatched(cusolverH, jobz, uplo, m, d_As, lda, d_W, d_work, lwork, d_info, syevj_params, batchSize);
		hipMemcpyAsync(V.data(), d_As, sizeof(float) * 16 * iters, hipMemcpyDeviceToHost, stream);
		hipMemcpyAsync(W.data(), d_W, sizeof(float) * 4 * iters, hipMemcpyDeviceToHost, stream);
		hipStreamSynchronize(stream);
		/////////////////

		//int* d_info = nullptr;
		//cublasCreate(&cublasH);
		//cublasSgelsBatched(cublasH, cublasOperation_t::CUBLAS_OP_N, 3, 3, 1, d_As, 1, d_bs, 1, d_info, NULL, iters);

		countInliers <<< iters / threadNum, threadNum >>> (d_pts, d_As, d_counts, d_plane_params, pts.size() / 3, threshold);
		checkCudaCall(hipDeviceSynchronize());

		checkCudaCall(hipMemcpy(h_counts, d_counts, iters * sizeof(int), hipMemcpyDeviceToHost));
		checkCudaCall(hipMemcpy(h_plane_params, d_plane_params, 4 * sizeof(float) * iters, hipMemcpyDeviceToHost));

		for (int i = 0; i < iters; i++)
		{
			counts[i] = h_counts[i];
			plane_params[i * 4] = h_plane_params[i * 4];
			plane_params[i * 4 + 1] = h_plane_params[i * 4 + 1];
			plane_params[i * 4 + 2] = h_plane_params[i * 4 + 2];
			plane_params[i * 4 + 3] = h_plane_params[i * 4 + 3];
		}
		
		checkCudaCall(hipGetLastError());

		checkCudaCall(hipDeviceSynchronize());
	}
	catch (CudaException ex)
	{
		goto Error;
	}
	
Error:
	cudaStatus = hipGetLastError();

	if(cublasH != NULL)
		hipblasDestroy(cublasH);

	if(d_pts != 0)
		hipFree(d_pts);

	if (d_plane_params != 0)
		hipFree(d_plane_params);

	if (d_counts != 0)
		hipFree(d_counts);

	hipFree(d_As);
	hipFree(d_bs);
	
	if (d_state != 0)
		hipFree(d_state);

	free(h_bs);
	free(h_As);
	free(h_plane_params);
	free(h_counts);

	return cudaStatus;
}