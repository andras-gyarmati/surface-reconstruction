#include "hip/hip_runtime.h"
#include "file_loader.h"
#include "VertexSet.h"
#define EIGEN_NO_CUDA
#include "Eigen/Dense"
#include "hip/hip_runtime.h"
#include ""
#include "hipsolver.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include <exception>
#include <random>

struct RANSACDiffs {
    int inliersNum;
    std::vector<bool> isInliers;
    std::vector<float> distances;
};

#pragma region GPU

class CudaException : public std::exception
{
	virtual const char* what() const throw()
	{
		return "GPUassert fail";
	}
};

#define checkCudaCall(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		throw CudaException();
	}
}

__device__ float distance(float* model, float* p)
{
	return fabsf(model[0] * p[0] + model[1] * p[1] + model[2] * p[2] + model[3]);
}

__global__ void countInliers(float* pts, float* eigenvectors, int* count_inliers, float* plane_params, int size, float threshold)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	float A = eigenvectors[id * 16];
	float B = eigenvectors[id * 16 + 1];
	float C = eigenvectors[id * 16 + 2];
	float D = eigenvectors[id * 16 + 3];

	float norm = sqrtf(A * A + B * B + C * C);

	plane_params[id * 4] = A / norm;
	plane_params[id * 4 + 1] = B / norm;
	plane_params[id * 4 + 2] = C / norm;
	plane_params[id * 4 + 3] = D / norm;

	float model[4] = { plane_params[id * 4] , plane_params[id * 4 + 1] , plane_params[id * 4 + 2], plane_params[id * 4 + 3] };

	int count = 0;
	for (int i = 0; i < size; i++)
	{
		float p[3] = { pts[i * 3], pts[i * 3 + 1], pts[i * 3 + 2] };
		if (distance(model, p) < threshold)
			count++;
	}

	count_inliers[id] = count;
}

__global__ void createSystems(float* pts, float* As, int seed, hiprandState* rand_state, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	hiprand_init(seed, id, 0, &rand_state[id]);
	int idx1, idx2, idx3;
	idx1 = hiprand_uniform(&(rand_state[id])) * size - 1;	//hiprand excludes 0.0 and includes 1.0
	idx2 = hiprand_uniform(&(rand_state[id])) * size - 1;
	idx3 = hiprand_uniform(&(rand_state[id])) * size - 1;

	float Aorig[3][4];
	Aorig[0][0] = pts[idx1 * 3];
	Aorig[0][1] = pts[idx1 * 3 + 1];
	Aorig[0][2] = pts[idx1 * 3 + 2];
	Aorig[0][3] = 1;
	Aorig[1][0] = pts[idx2 * 3];
	Aorig[1][1] = pts[idx2 * 3 + 1];
	Aorig[1][2] = pts[idx2 * 3 + 2];
	Aorig[1][3] = 1;
	Aorig[2][0] = pts[idx3 * 3];
	Aorig[2][1] = pts[idx3 * 3 + 1];
	Aorig[2][2] = pts[idx3 * 3 + 2];
	Aorig[2][3] = 1;

	float At[4][3];
	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 3; j++)
		{
			At[i][j] = Aorig[j][i];
		}
	}

	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			As[id * 16 + j * 4 + i] = 0;

			for (int k = 0; k < 3; k++) {
				//Column major ordering
				As[id * 16 + j * 4 + i] += At[i][k] * Aorig[k][j];
			}
		}
	}
}

hipError_t runRansacGPU(const std::vector<float>& pts, std::vector<float>& plane_params, std::vector<int>& counts, const int& iters, const float& threshold, const int& threadNum)
{
	float* d_pts = 0;
	float* d_plane_params = 0;
	int* d_counts = 0;
	float* d_As = 0;

	int* h_counts = (int*)malloc(sizeof(int) * iters);
	float* h_plane_params = (float*)malloc(sizeof(float) * 4 * iters);

	hiprandState* d_state;
	hipError_t cudaStatus;

	hipblasHandle_t cublasH = NULL;

	try {
		checkCudaCall(hipSetDevice(0));

		checkCudaCall(hipMalloc((void**)&d_pts, pts.size() * sizeof(float)));
		checkCudaCall(hipMalloc((void**)&d_plane_params, iters * 4 * sizeof(float)));
		checkCudaCall(hipMalloc((void**)&d_counts, iters * sizeof(float)));

		checkCudaCall(hipMalloc(&d_state, iters * sizeof(hiprandState)));
		checkCudaCall(hipMalloc((void**)&d_As, iters * 16 * sizeof(float)));

		checkCudaCall(hipMemcpy(d_pts, pts.data(), pts.size() * sizeof(float), hipMemcpyHostToDevice));

		srand(time(0));
		int seed = rand();

		createSystems <<< iters / threadNum, threadNum >>> (d_pts, d_As, seed, d_state, pts.size() / 3);

		//////////////////
		hipsolverHandle_t cusolverH = NULL;
		hipStream_t stream = NULL;
		hipsolverSyevjInfo_t syevj_params = NULL;

		const int m = 4;
		const int lda = 4;
		const int batchSize = iters;

		std::vector<float> V(lda * m * iters, 0);
		std::vector<float> W(m * iters, 0);
		std::vector<int> info(batchSize, 0);

		float* d_W = nullptr;
		int* d_info = nullptr;
		float* d_work = nullptr;
		int lwork = 0;

		/* configuration of syevj  */
		const double tol = 1.e-7;
		const int max_sweeps = 15;
		const int sort_eig = 1;                           /* sort eigenvalues/vectors in ascending order */
		const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; /* compute eigenvectors */
		const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

		hipsolverDnCreate(&cusolverH);
		hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
		hipsolverSetStream(cusolverH, stream);
		hipsolverDnCreateSyevjInfo(&syevj_params);
		hipsolverDnXsyevjSetTolerance(syevj_params, tol);
		hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps);
		hipsolverDnXsyevjSetSortEig(syevj_params, sort_eig);

		hipMalloc((void**)&d_W, sizeof(float) * W.size());
		hipMalloc((void**)&d_info, sizeof(int) * info.size());
		hipsolverDnSsyevjBatched_bufferSize(cusolverH, jobz, uplo, m, d_As, lda, d_W, &lwork, syevj_params, batchSize);
		hipMalloc(reinterpret_cast<void**>(&d_work), sizeof(double) * lwork);
		hipsolverDnSsyevjBatched(cusolverH, jobz, uplo, m, d_As, lda, d_W, d_work, lwork, d_info, syevj_params, batchSize);
		hipMemcpyAsync(V.data(), d_As, sizeof(float) * 16 * iters, hipMemcpyDeviceToHost, stream);
		hipMemcpyAsync(W.data(), d_W, sizeof(float) * 4 * iters, hipMemcpyDeviceToHost, stream);
		hipStreamSynchronize(stream);
		/////////////////

		countInliers <<< iters / threadNum, threadNum >>> (d_pts, d_As, d_counts, d_plane_params, pts.size() / 3, threshold);
		checkCudaCall(hipDeviceSynchronize());

		checkCudaCall(hipMemcpy(h_counts, d_counts, iters * sizeof(int), hipMemcpyDeviceToHost));
		checkCudaCall(hipMemcpy(h_plane_params, d_plane_params, 4 * sizeof(float) * iters, hipMemcpyDeviceToHost));

		for (int i = 0; i < iters; i++)
		{
			counts[i] = h_counts[i];
			plane_params[i * 4] = h_plane_params[i * 4];
			plane_params[i * 4 + 1] = h_plane_params[i * 4 + 1];
			plane_params[i * 4 + 2] = h_plane_params[i * 4 + 2];
			plane_params[i * 4 + 3] = h_plane_params[i * 4 + 3];
		}

		checkCudaCall(hipGetLastError());

		checkCudaCall(hipDeviceSynchronize());
	}
	catch (CudaException ex)
	{
		goto Error;
	}

Error:
	cudaStatus = hipGetLastError();

	if (cublasH != NULL)
		hipblasDestroy(cublasH);

	if (d_pts != 0)
		hipFree(d_pts);

	if (d_plane_params != 0)
		hipFree(d_plane_params);

	if (d_counts != 0)
		hipFree(d_counts);

	if (d_As != 0)
		hipFree(d_As);

	if (d_state != 0)
		hipFree(d_state);

	free(h_plane_params);
	free(h_counts);

	return cudaStatus;
}

#pragma endregion


#pragma region CPU

float* EstimatePlaneImplicit(const VertexSet& pts, const std::vector<int>& indices) {
    const size_t num = indices.size();

    Eigen::MatrixXf Cfs(num, 4);

    for (size_t i = 0; i < num; i++) {
        file_loader::vertex pt = pts[indices[i]];
        Cfs(i, 0) = pt.position.x;
        Cfs(i, 1) = pt.position.y;
        Cfs(i, 2) = pt.position.z;
        Cfs(i, 3) = 1.0f;
    }

    Eigen::MatrixXf mtx = Cfs.transpose() * Cfs;
    Eigen::EigenSolver<Eigen::MatrixXf> es(mtx);

    const int lowestEigenValueIndex = std::min({ 0,1,2,3 },
        [&es](int v1, int v2) {
            return es.eigenvalues()[v1].real() < es.eigenvalues()[v2].real();
        });

    float A = es.eigenvectors().col(lowestEigenValueIndex)(0).real();
    float B = es.eigenvectors().col(lowestEigenValueIndex)(1).real();
    float C = es.eigenvectors().col(lowestEigenValueIndex)(2).real();
    float D = es.eigenvectors().col(lowestEigenValueIndex)(3).real();

    float norm = std::sqrt(A * A + B * B + C * C);

    float* ret = new float[4];
    ret[0] = A / norm;
    ret[1] = B / norm;
    ret[2] = C / norm;
    ret[3] = D / norm;

    return ret;
}

RANSACDiffs PlanePointRANSACDifferences(const VertexSet& pts, const std::vector<int>& indices, float* plane, float threshold) {
    size_t num = indices.size();

    float A = plane[0];
    float B = plane[1];
    float C = plane[2];
    float D = plane[3];

    RANSACDiffs ret;

    std::vector<bool> isInliers;
    std::vector<float> distances;

    int inlierCounter = 0;
    for (int idx = 0; idx < num; idx++) {
        file_loader::vertex pt = pts[indices[idx]];
        float diff = fabs(A * pt.position.x + B * pt.position.y + C * pt.position.z + D);
        distances.push_back(diff);
        if (diff < threshold) {
            isInliers.push_back(true);
            ++inlierCounter;
        }
        else {
            isInliers.push_back(false);
        }
    }

    ret.distances = distances;
    ret.isInliers = isInliers;
    ret.inliersNum = inlierCounter;

    return ret;
}

float* EstimatePlaneRANSAC(const VertexSet& pts, const std::vector<int>& indices, float threshold, int iterNum) {
    size_t num = indices.size();

    int bestSampleInlierNum = 0;
    float* bestPlane = new float[4];

    for (size_t iter = 0; iter < iterNum; iter++) {
        int index1 = rand() % num;
        int index2 = rand() % num;

        while (index2 == index1) {
            index2 = rand() % num;
        }
        int index3 = rand() % num;
        while (index3 == index1 || index3 == index2) {
            index3 = rand() % num;
        }

        const std::vector<int> minimalSample = { index1, index2, index3 };

        float* samplePlane = EstimatePlaneImplicit(pts, minimalSample);

        RANSACDiffs sampleResult = PlanePointRANSACDifferences(pts, indices, samplePlane, threshold);

        if (sampleResult.inliersNum > bestSampleInlierNum) {
            bestSampleInlierNum = sampleResult.inliersNum;
            for (int i = 0; i < 4; ++i) {
                bestPlane[i] = samplePlane[i];
            }
        }

        delete[] samplePlane;
    }

    RANSACDiffs bestResult = PlanePointRANSACDifferences(pts, indices, bestPlane, threshold);
    std::cout << "Best plane params: " << bestPlane[0] << " " << bestPlane[1] << " " << bestPlane[2] << "\n";

    std::vector<int> inlierPts;

    for (int idx = 0; idx < num; idx++) {
        if (bestResult.isInliers.at(idx)) {
            inlierPts.push_back(indices[idx]);
        }
    }

    float* finalPlane = EstimatePlaneImplicit(pts, inlierPts);
    delete[] bestPlane;
    return finalPlane;
}

#pragma endregion

RANSACDiffs runRANSACPlane(const VertexSet& pts, const std::vector<int> indices, const int& iters, const float& threshold)
{
    float* bestModel;

#ifdef GPU

    std::vector<float> input(indices.size() * 3);
    std::vector<float> plane_params(iters * 4);
    std::vector<int> count(iters);

    for (size_t j = 0; j < indices.size(); j++)
    {
        input[j * 3] = pts[indices[j]].position.x;
        input[j * 3 + 1] = pts[indices[j]].position.y;
        input[j * 3 + 2] = pts[indices[j]].position.z;

        //std::cout << m_vertices[filteredPoints[j]].position.x << " " << m_vertices[filteredPoints[j]].position.y << m_vertices[filteredPoints[j]].position.z << "\n";
    }

    hipError_t cudaStatus = runRansacGPU(input, plane_params, count, iters, threshold, 256);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "runRansacGPU failed!");
    }

    int mostInlierIndex = 0;
    for (int i = 0; i < iters; i++)
    {
        if (count[i] > count[mostInlierIndex])
            mostInlierIndex = i;
    }

    std::cout << count[mostInlierIndex] << "\n";

    bestModel = new float[4];
    bestModel[0] = plane_params[mostInlierIndex * 4];
    bestModel[1] = plane_params[mostInlierIndex * 4 + 1];
    bestModel[2] = plane_params[mostInlierIndex * 4 + 2];
    bestModel[3] = plane_params[mostInlierIndex * 4 + 3];

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
    }

#endif // GPU

#ifndef GPU

    bestModel = EstimatePlaneRANSAC(pts, indices, threshold, iters);

#endif // !GPU

    size_t num = indices.size();

    std::cout << "Plane params RANSAC:" << std::endl;
    std::cout << "A:" << bestModel[0] << " B:" << bestModel[1]
        << " C:" << bestModel[2] << " D:" << bestModel[3] << std::endl;

    RANSACDiffs differences = PlanePointRANSACDifferences(pts ,indices, bestModel, threshold);
    delete[] bestModel;

    return differences;
}